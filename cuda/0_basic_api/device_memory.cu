#include "hip/hip_runtime.h"

#include <stdio.h>

void checkDeviceMemory(void)
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    // free : 가용 메모리(사용 가능한 메모리), total : 총 디바이스 메모리
    printf("Device memory (free/total) = %lld/%lld bytes\n", free, total);
}

int main(void)
{
    int* dDataPtr;
    hipError_t errorCode;

    checkDeviceMemory();
    // 디바이스 메모리 할당
    errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024);
    printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();

    // 디바이스 메모리 초기화
    errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024);
    printf("hipMemset - %s\n", hipGetErrorName(errorCode));

    // 메모리 해제
    errorCode = hipFree(dDataPtr);
    printf("hipFree - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();

}
