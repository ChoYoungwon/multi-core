#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void helloCUDA(void)
{
    printf("Hello CUDA from GPU!\n");
}

int main(void)
{
    printf("Hello GPU from CPU!\n");
    helloCUDA<<<1,10>>>();

    // GPU 작업이 끝날 때까지 대기
    hipDeviceSynchronize();
    
    return 0;
}